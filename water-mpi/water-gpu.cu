#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

extern float toBW(int bytes, float sec);


/* Helper function to round up to a power of 2. 
 */
static inline int nextPow2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}


__global__ void
exclusive_scan_downsweep_kernel(int* device_data, int length, int twod, int twod1) {
    int i = (blockIdx.x * blockDim.x + threadIdx.x) * twod1;
    int t_index = i + twod - 1;
    int second_index = i + twod1 - 1;

    if (i < length && t_index < length && second_index < length) {
        int t = device_data[t_index];
        device_data[t_index] = device_data[second_index];
        device_data[second_index] += t;
    }
}

__global__ void
exclusive_scan_upsweep_kernel(int* device_data, int length, int twod, int twod1) {
    int i = (blockIdx.x * blockDim.x + threadIdx.x) * twod1;
    int index_add_val = i + twod - 1;
    int index_add_to = i + twod1 - 1;

    if (i < length && index_add_val < length && index_add_to < length)
        device_data[index_add_to] += device_data[index_add_val];
}

__global__ void
set_last_to_zero(int * device_data, int N) {
    device_data[N-1] = 0;
}

__global__ void
set_peeks(int * y_vals, int * peek_sig, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i == 0) peek_sig[i] = 0;
    if (i == N - 1) peek_sig[i] = 0;
    if (1 <= i && i < N - 1) {
      if (y_vals[i - 1] < y_vals[i] && y_vals[i] > y_vals[i+1])
        peek_sig[i] = 1;
      else peek_sig[i] = 0;
    }
}

__global__ void
write_peeks(int * peek_sigs_summed, int * peek_idxs, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (1 <= i && i < N && peek_sigs_summed[i] != peek_sigs_summed[i - 1]){
        peek_idxs[peek_sigs_summed[i - 1]] = i - 1;
    }
}

void exclusive_scan(int* device_data, int length)
{
    /* TODO
     * Fill in this function with your exclusive scan implementation.
     * You are passed the locations of the data in device memory
     * The data are initialized to the inputs.  Your code should
     * do an in-place scan, generating the results in the same array.
     * This is host code -- you will need to declare one or more CUDA 
     * kernels (with the __global__ decorator) in order to actually run code
     * in parallel on the GPU.
     * Note you are given the real length of the array, but may assume that
     * both the data array is sized to accommodate the next
     * power of 2 larger than the input.
     */
    const int threadsPerBlock = 512;
    const int N = nextPow2(length);

    for (int twod = 1; twod < N; twod*=2)
    {
        int twod1 = twod * 2;
        // compute number of blocks and threads per block
        int blocks = ((length / twod1) + 1 + threadsPerBlock - 1) / threadsPerBlock;
        exclusive_scan_upsweep_kernel<<<blocks, threadsPerBlock>>>(
           device_data, N, twod, twod1);
        hipDeviceSynchronize();
    }
    set_last_to_zero<<<1, 1>>>(device_data, N);
    hipDeviceSynchronize();
    // downsweep phase.
    for (int twod = N/2; twod >= 1; twod /= 2)
    {
        int twod1 = twod * 2;
        int blocks = (length + (threadsPerBlock * twod1) - 1) / (threadsPerBlock * twod1);
        exclusive_scan_downsweep_kernel<<<blocks, threadsPerBlock>>>(
            device_data, N, twod, twod1);
        hipDeviceSynchronize();
    }
}

/* This function is a wrapper around the code you will write - it copies the
 * input to the GPU and times the invocation of the exclusive_scan() function
 * above. You should not modify it.
 */
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_data;
    // We round the array size up to a power of 2, but elements after
    // the end of the original input are left uninitialized and not checked
    // for correctness. 
    // You may have an easier time in your implementation if you assume the 
    // array's length is a power of 2, but this will result in extra work on
    // non-power-of-2 inputs.
    int rounded_length = nextPow2(end - inarray);
    hipMalloc((void **)&device_data, sizeof(int) * rounded_length);

    hipMemcpy(device_data, inarray, (end - inarray) * sizeof(int), 
               hipMemcpyHostToDevice);


    exclusive_scan(device_data, end - inarray);

    // Wait for any work left over to be completed.
    hipDeviceSynchronize();
    double overallDuration = 0;
    
    hipMemcpy(resultarray, device_data, (end - inarray) * sizeof(int),
               hipMemcpyDeviceToHost);
    return overallDuration;
}

/* Wrapper around the Thrust library's exclusive scan function
 * As above, copies the input onto the GPU and times only the execution
 * of the scan itself.
 * You are not expected to produce competitive performance to the
 * Thrust version.
 */
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), 
               hipMemcpyHostToDevice);


    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int),
               hipMemcpyDeviceToHost);
    thrust::device_free(d_input);
    thrust::device_free(d_output);
    double overallDuration = 0;
    return overallDuration;
}



int find_peaks(int *device_input, int length, int *device_output) {
    /* TODO:
     * Finds all elements in the list that are greater than the elements before and after,
     * storing the index of the element into device_result.
     * Returns the number of peak elements found.
     * By definition, neither element 0 nor element length-1 is a peak.
     *
     * Your task is to implement this function. You will probably want to
     * make use of one or more calls to exclusive_scan(), as well as
     * additional CUDA kernel launches.
     * Note: As in the scan code, we ensure that allocated arrays are a power
     * of 2 in size, so you can use your exclusive_scan function with them if 
     * it requires that. However, you must ensure that the results of
     * find_peaks are correct given the original length.
     */    
     // The algorithm consists of setting peeks to 1 and non-peeks to 0.
     // The inclusive scan and remove the duplicates
    const int threadsPerBlock = 512;
    const int blocks = (length + threadsPerBlock - 1) / threadsPerBlock;
    int * peek_sig;
    hipMalloc(&peek_sig, sizeof(int) * length);

    set_peeks<<<blocks, threadsPerBlock>>>(device_input, peek_sig, length);
    hipDeviceSynchronize();
    int * peek_sigs_summed = peek_sig;

    exclusive_scan(peek_sigs_summed, length);
    hipDeviceSynchronize();

    int * numb_peeks = (int *) malloc(sizeof(int));
    hipMemcpy(numb_peeks, &peek_sigs_summed[length - 1], sizeof(int), hipMemcpyDeviceToHost);

    write_peeks<<<blocks, threadsPerBlock>>>(peek_sigs_summed, device_output, length);
    hipDeviceSynchronize();

    int numb_peeks_tmp = numb_peeks[0];
    free(numb_peeks);
    hipFree(peek_sig);

    return numb_peeks_tmp;
}



/* Timing wrapper around find_peaks. You should not modify this function.
 */
double cudaFindPeaks(int *input, int length, int *output, int *output_length) {
    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), 
               hipMemcpyHostToDevice);

    
    int result = find_peaks(device_input, length, device_output);

    hipDeviceSynchronize();

    *output_length = result;

    hipMemcpy(output, device_output, length * sizeof(int),
               hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    return 0;
}


void computeSection()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}